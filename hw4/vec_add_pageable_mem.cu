#include "hip/hip_runtime.h"
#include <stdio.h>

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  //这里使用stride加循环的方法来处理数组大小不是block倍数的情况
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
    const unsigned int startTime = clock();

    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    const int N = 1e8;
    size_t size = N * sizeof(float);

    float *a, *dev_a;
    float *b, *dev_b;
    float *c, *dev_c;

    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, size);

    a = (float*)malloc(size);
    b = (float*)malloc(size);
    c = (float*)malloc(size);

    initWith(3, a, N);
    initWith(4, b, N);
    initWith(0, c, N);

    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, size, hipMemcpyHostToDevice);

    //   hipMemPrefetchAsync(a, size, deviceId);
    //   hipMemPrefetchAsync(b, size, deviceId);
    //   hipMemPrefetchAsync(c, size, deviceId);

    size_t threadsPerBlock;
    size_t numberOfBlocks;

    threadsPerBlock = 256;
    numberOfBlocks = 32 * numberOfSMs;

    hipError_t addVectorsErr;
    hipError_t asyncErr;

    addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(dev_c, dev_a, dev_b, N);

    addVectorsErr = hipGetLastError();
    if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

    asyncErr = hipDeviceSynchronize();
    if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

    checkElementsAre(7, c, N);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    free(a);
    free(b);
    free(c);

    const unsigned int endTime = clock();

    printf("time cost : %d\n", endTime - startTime);
}